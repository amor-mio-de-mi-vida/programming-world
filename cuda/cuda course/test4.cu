#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>

using namespace std;
using namespace cv;

// Sobel:
// kernel   x00 x01 x02      m00 m01 m02 m03 m04        r00 r01 r02 r03 r04
//          x10 x11 x12      m10 m11 m12 m13 m14        r10 r11 r12 r13 r14
//          x20 x21 x22      m20 m21 m22 m23 m24        r20 r21 r22 r23 r24
//                           m30 m31 m32 m33 m34        r30 r31 r32 r33 r34
//                           m40 m41 m42 m43 m44        r40 r41 r42 r43 r44
//
//  r11 = x00*m00 + x01*m01 + x02*m02 + x10*m10 + x11*m11 + x12*m12 + x20*m20 + x21*m21 + x22*m22
//  Gx  1 0 -1      Gy   1  2  1
//      2 0 -2           0  0  0
//      1 0 -1          -1 -2 -1

__global__ void sobel_gpu(unsigned char* in, unsigned char* out, const int Height, constint Width);

int main() {
    Mat img = imread("1.jpg", 0);
    int height = img.rows;
    int width = img.cols;

    Mat gaussImg;
    GaussianBlur(img, gaussImg, Size(3,3), 0, 0, BORDER_DEFAULT);

    Mat dst_gpu(height, width, CV_8UC1, Scalar(0));

    int memsize = height * width * sizeof(unsigned char);

    unsigned char* in_gpu;
    unsigned char* out_gpu;

    hipMalloc((void**) &in_gpu, memsize);
    hipMalloc((void**) &out_gpu, memsize);

    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((width + threadPerBlock.x - 1) / threadPerBlock.x , (height + threadPerBlock.y -1) / threadPerBlock.y);

    hipMemcpy(in_gpu, gaussImg.data, memsize, hipMemcpyHostToDevice);

    sobel_gpu<<<blocksPerGrid, threadsPerBlock>>>(in_gpu, out_gpu, height, width);

    hipMemcpy(dst_gpu.data, out_gpu, memsize, hipMemcpyDeviceToHost);

    imwrite("save.png", dst_gpu);
    hipFree(in_gpu);
    hipFree(out_gpu);

    return0;
}

__global__ void sobel_gpu(unsigned char* in, unsigned char* out, const int Height, constint Width) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int index = y * Width + x;

    int Gx = 0;
    int Gy = 0;
    unsigned char x0, x1, x2, x3, x4, x5, x6, x7, x8;
    // SM register
    if (x > 0 && x < (Width - 1) && y > 0 && y < (Height - 1)) {
        x0 = in[(y-1) * Width + (x-1)];
        x1 = in[(y-1) * Width + x];
        x2 = in[(y-1) * Width + (x+1)];
        x3 = in[(y) * Width + (x-1)];
        x4 = in[(y) * Width + x];
        x5 = in[(y) * Width + (x+1)];
        x6 = in[(y+1) * Width + (x-1)];
        x7 = in[(y+1) * Width + x];
        x8 = in[(y+1) * Width + (x+1)];

        Gx = (x0 + 2*x3 + x6) - (x2 + 2*x5 + x7);
        Gy = (x0 + 2*x1 + x2) - (x6 + 2*x7 + x8);
        
        out[index] = (Gx + Gy) / 2;
    }

}